
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__device__ void function(int *a,int b)
{
    a[0] = a[0]^b;
    return;
}
__global__ void calculate(int *mem, int num,int skip)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i%(2*skip)!=0) return;
      if(i+skip >= num)
           return;
       int *a = mem+i;
      function(a,a[skip]);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    int *ptr;
    int *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks,seed;

         num = atoi(argv[1]);   /*Update after checking*/
	seed = atoi(argv[2]);
         if(num <= 0)
               num = NUM;
	if(num>=(1<<20)) {
		printf("maxlimit is set to 2^20\n");
		return 0;
	}
	srand(seed);
    /* Allocate host (CPU) memory and initialize*/

    ptr = (int *)malloc(num  * sizeof(int));
    for(i=0; i<num; ++i){
       ptr[i] = rand();
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num* sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");
    hipMemcpy(gpu_mem, ptr, num  * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
   blocks = num /1024;
    
    if(num % 1024)
           ++blocks;
	for(int i = 0 ; i <21;i++){
	 calculate<<<blocks,1024>>>(gpu_mem, num,1<<i);
	}
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num  * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);

	printf("%d\n",ptr[0]);
     
    free(ptr);
}
